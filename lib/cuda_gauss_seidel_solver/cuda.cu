#include "hip/hip_runtime.h"
#include <cstddef>
#include <array>
#include <vector>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>

#include "clion_cuda_stub.h"
#include "cuda_gauss_seidel_solver.h"


namespace cuda_gauss_seidel_solver {
	struct MatrixRow {
		unsigned int indices[4];
	};

	template <typename RealT>
	struct SolutionRow {
		RealT values[2];
	};

#define HANDLE_ERROR(err) (cuda_handle_error((err), __FILE__, __LINE__ ))

	__host__ void cuda_handle_error(hipError_t err, const char* file, int line) {
		if (err != hipSuccess) {
			printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
			throw std::runtime_error("CUDA error");
		}
	}

	template <typename RealT>
	__global__ void _kernel_gauss_seidel(const MatrixRow* mat,
										 unsigned int mat_size,
										 const int* rhs,
										 RealT* old_solution,
										 RealT* solution) {
		auto thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (thread_idx >= mat_size)
			return;

		RealT row_res = rhs[thread_idx];
		#pragma unroll 4
		for (int i = 0; i < 4; i++) {
			row_res -= old_solution[mat[thread_idx].indices[i]];
		}

		solution[thread_idx] = row_res / -4;
	}

	template <typename T>
	T ceil_div(T a, T d) {
		return (a + d - 1) / d;
	}

	__host__ void gauss_seidel_solve(
			const std::vector<std::array<int, 4>>& mat,
			std::array<const std::vector<int>, 3>& rhs,
			std::array<std::vector<real_t>, 3>& sol,
			unsigned int iterations) {
		constexpr unsigned max_threads_per_block = 1024;

		MatrixRow* d_mat;
		std::array<int*, 3> d_rhs;
		std::array<std::array<real_t*, 2>, 3> d_sol;
		unsigned mat_size = (unsigned) mat.size();

		unsigned block_size = 128;
		unsigned block_count = ceil_div(mat_size, block_size);

		static_assert(sizeof(*d_mat) == sizeof(mat[0]), "Device matrix use different datatype");
		HANDLE_ERROR(hipMalloc((void**) &d_mat, mat.size() * sizeof(*d_mat)));
		HANDLE_ERROR(hipMemcpy(d_mat, mat.data(),
								mat_size * sizeof(*d_mat),
								hipMemcpyHostToDevice));

		for (int i = 0; i < 3; i++) {
			HANDLE_ERROR(hipMalloc((void**) &(d_rhs[i]), rhs[i].size() * sizeof(int)));
			HANDLE_ERROR(hipMalloc((void**) &(d_sol[i][0]), sol[i].size() * sizeof(real_t)));
			HANDLE_ERROR(hipMalloc((void**) &(d_sol[i][1]), sol[i].size() * sizeof(real_t)));

			HANDLE_ERROR(hipMemcpy(d_rhs[i], rhs[i].data(),
									rhs[i].size() * sizeof(int),
									hipMemcpyHostToDevice));

			HANDLE_ERROR(hipMemcpy(d_sol[i][0], sol[i].data(),
									sol[i].size() * sizeof(real_t),
									hipMemcpyHostToDevice));
		}

		std::array<hipStream_t, 3> streams;
		for (int i = 0; i < 3; i++) {
			HANDLE_ERROR(hipStreamCreate(&streams[i]));
		}

		int buf_idx = 0;
		for (int it = 0; it < iterations; it++) {
			for (int i = 0; i < 3; i++)
				_kernel_gauss_seidel<<<block_count, block_size, 0, streams[i]>>> (
						d_mat, mat_size, d_rhs[i], d_sol[i][buf_idx], d_sol[i][buf_idx ^ 1]);
			buf_idx ^= 1;
		}

		HANDLE_ERROR(hipDeviceSynchronize());

		for (int i = 0; i < 3; i++) {
			HANDLE_ERROR(hipStreamDestroy(streams[i]));
		}

		for (int i = 0; i < 3; i++) {
			HANDLE_ERROR(hipMemcpy(sol[i].data(), d_sol[i][buf_idx],
									sol[i].size() * sizeof(real_t),
									hipMemcpyDeviceToHost));

			HANDLE_ERROR(hipFree(d_sol[i][1]));
			HANDLE_ERROR(hipFree(d_sol[i][0]));
			HANDLE_ERROR(hipFree(d_rhs[i]));
		}
		HANDLE_ERROR(hipFree(d_mat));
	}
}
