#include "hip/hip_runtime.h"
#include <cstddef>
#include <array>
#include <vector>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>

#include "clion_cuda_stub.h"
#include "cuda_gauss_seidel_solver.h"


namespace cuda_gauss_seidel_solver {
	struct MatrixRow {
		unsigned int indices[4];
	};

	template <typename RealT>
	struct SolutionRow {
		RealT values[2];
	};

#define HANDLE_ERROR(err) (cuda_handle_error((err), __FILE__, __LINE__ ))

	__host__ void cuda_handle_error(hipError_t err, const char* file, int line) {
		if (err != hipSuccess) {
			printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
			throw std::runtime_error("CUDA error");
		}
	}

	template <typename RealT>
	__global__ void _kernel_gauss_seidel(const MatrixRow* mat,
										 unsigned int mat_size,
										 const int* rhs,
										 RealT* old_solution,
										 RealT* solution) {
		auto thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (thread_idx >= mat_size)
			return;

		RealT row_res = rhs[thread_idx];
		#pragma unroll 4
		for (int i = 0; i < 4; i++) {
			row_res -= old_solution[mat[thread_idx].indices[i]];
		}

		solution[thread_idx] = row_res / -4;
	}

	template <typename T>
	T ceil_div(T a, T d) {
		return (a + d - 1) / d;
	}

	__host__ void gauss_seidel_solve(
			const std::vector<std::array<int, 4>>& mat,
			const std::vector<int>& rhs,
			std::vector<real_t>& sol,
			unsigned int iterations) {
		constexpr unsigned max_threads_per_block = 1024;

		MatrixRow* d_mat;
		int* d_rhs;
		real_t* d_sol[2];
		unsigned mat_size = (unsigned) mat.size();

		unsigned block_size = 128;
		unsigned block_count = ceil_div(mat_size, block_size);

		static_assert(sizeof(*d_mat) == sizeof(mat[0]), "Device matrix use different datatype");
		HANDLE_ERROR(hipMalloc((void**) &d_mat, mat.size() * sizeof(*d_mat)));
		HANDLE_ERROR(hipMalloc((void**) &d_rhs, rhs.size() * sizeof(*d_rhs)));
		HANDLE_ERROR(hipMalloc((void**) d_sol, sol.size() * sizeof(real_t)));
		HANDLE_ERROR(hipMalloc((void**) (d_sol + 1), sol.size() * sizeof(real_t)));

		HANDLE_ERROR(hipMemcpy(d_mat, mat.data(),
								mat_size * sizeof(*d_mat),
								hipMemcpyHostToDevice));

		HANDLE_ERROR(hipMemcpy(d_rhs, rhs.data(),
								rhs.size() * sizeof(*d_rhs),
								hipMemcpyHostToDevice));

		HANDLE_ERROR(hipMemcpy(d_sol[0], sol.data(),
								sol.size() * sizeof(real_t),
								hipMemcpyHostToDevice));

		int buf_idx = 0;
		for (int i = 0; i < iterations; i++) {
			_kernel_gauss_seidel <<<block_count, block_size>>> (
					d_mat, mat_size, d_rhs, d_sol[buf_idx], d_sol[buf_idx ^ 1]);
			buf_idx ^= 1;
		}

		HANDLE_ERROR(hipDeviceSynchronize());

		HANDLE_ERROR(hipMemcpy(sol.data(), d_sol[buf_idx],
								sol.size() * sizeof(real_t),
								hipMemcpyDeviceToHost));

		HANDLE_ERROR(hipFree(d_mat));
		HANDLE_ERROR(hipFree(d_rhs));
		HANDLE_ERROR(hipFree(d_sol[0]));
		HANDLE_ERROR(hipFree(d_sol[1]));
	}
}
