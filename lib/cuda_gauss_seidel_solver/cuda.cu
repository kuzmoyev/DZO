#include "hip/hip_runtime.h"
#include <cstddef>
#include <array>
#include <vector>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>

#include "clion_cuda_stub.h"
#include "cuda_gauss_seidel_solver.h"


namespace cuda_gauss_seidel_solver {
	struct MatrixRow {
		unsigned int indices[4];
	};

	template <typename RealT>
	struct SolutionRow {
		RealT values[2];
	};

#define HANDLE_ERROR(err) (cuda_handle_error((err), __FILE__, __LINE__ ))

	__host__ void cuda_handle_error(hipError_t err, const char* file, int line) {
		if (err != hipSuccess) {
			printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
			throw std::runtime_error("CUDA error");
		}
	}

	template <typename RealT>
	__global__ void _kernel_gauss_seidel(const MatrixRow* mat, unsigned int mat_size,
										 const int* rhs, SolutionRow<RealT>* solution,
										 unsigned int iterations,
										 unsigned int items_per_thread) {
		int swap_idx = 0;
		#define IDX_OLD (swap_idx)
		#define IDX_NEW (swap_idx ^ 1)
		#define SWAP_BUFFERS swap_idx ^= 1

		auto index_from = threadIdx.x * items_per_thread;
		auto index_to = index_from + items_per_thread;
		if (index_to > mat_size)
			index_to = mat_size;

		for (unsigned int it = 0; it < iterations; it++) {
			for (size_t row = index_from; row < index_to; row++) {
				RealT row_res = rhs[row];
				#pragma unroll 4
				for (int i = 0; i < 4; i++) {
					row_res -= solution[mat[row].indices[i]].values[IDX_OLD];
				}

				solution[row].values[IDX_NEW] = row_res / -4;
			}
			SWAP_BUFFERS;
			__syncthreads();
		}
	}

	template <typename T>
	T ceil_div(T a, T d) {
		return (a + d - 1) / d;
	}

	__host__ void gauss_seidel_solve(
			const std::vector<std::array<int, 4>>& mat,
			const std::vector<int>& rhs,
			std::vector<real_t>& sol,
			unsigned int iterations) {
		constexpr unsigned max_threads_per_block = 1024;

		MatrixRow* d_mat;
		int* d_rhs;
		SolutionRow<real_t>* d_sol;
		unsigned mat_size = (unsigned) mat.size();

		unsigned items_per_thread = ceil_div(mat_size, max_threads_per_block);

		static_assert(sizeof(*d_mat) == sizeof(mat[0]), "Device matrix use different datatype");
		HANDLE_ERROR(hipMalloc((void**) &d_mat, mat.size() * sizeof(*d_mat)));
		HANDLE_ERROR(hipMalloc((void**) &d_rhs, rhs.size() * sizeof(*d_rhs)));
		HANDLE_ERROR(hipMalloc((void**) &d_sol, sol.size() * sizeof(*d_sol)));

		HANDLE_ERROR(hipMemcpy(d_mat, mat.data(),
								mat_size * sizeof(*d_mat),
								hipMemcpyHostToDevice));

		HANDLE_ERROR(hipMemcpy(d_rhs, rhs.data(),
								rhs.size() * sizeof(*d_rhs),
								hipMemcpyHostToDevice));

		std::vector<SolutionRow<real_t>> doubled_solution(sol.size());
		for (size_t i = 0; i < sol.size(); i++)
			doubled_solution[i].values[0] = sol[i];

		HANDLE_ERROR(hipMemcpy(d_sol, doubled_solution.data(),
								doubled_solution.size() * sizeof(*d_sol),
								hipMemcpyHostToDevice));

		_kernel_gauss_seidel<<<1, 1024>>>(d_mat, mat_size, d_rhs, d_sol,
				iterations, items_per_thread);

		HANDLE_ERROR(hipDeviceSynchronize());

		HANDLE_ERROR(hipMemcpy(doubled_solution.data(), d_sol,
								doubled_solution.size() * sizeof(*d_sol),
								hipMemcpyDeviceToHost));

		if (iterations % 2 == 0) {
			for (size_t i = 0; i < sol.size(); i++)
				sol[i] = doubled_solution[i].values[0];
		} else {
			for (size_t i = 0; i < sol.size(); i++)
				sol[i] = doubled_solution[i].values[1];
		}

		HANDLE_ERROR(hipFree(d_mat));
		HANDLE_ERROR(hipFree(d_rhs));
		HANDLE_ERROR(hipFree(d_sol));
	}
}
